#include "hip/hip_runtime.h"
#include "utils.h"
#define BLOCKSPARSE_THREADS 32
#define BLOCKSPARSE_MAXOUTPUTBLOCKSIZE 512
#define BLOCKSPARSE_STREAMS 8
  
__global__ void cunnx_BlockSparse_updateOutput_kernel(
  float *output, const float *input, const float *outputIndice, 
  const float *outputScale, const float *bias,  
  int outputSize, int nOutputBlock, 
  int inputWindowSize, int outputWindowSize)
{
  __shared__ float buffer[BLOCKSPARSE_THREADS];
  int tx = threadIdx.x;
  int i_step = blockDim.x;
  int k = blockIdx.x;
  
  float *output_k = output + k*outputWindowSize*outputSize;
  const float *input_k = input + k*inputWindowSize*outputWindowSize*outputSize;
  const float *outputIndice_k = outputIndice + k*outputWindowSize;
  const float *outputScale_k = outputScale + k*outputWindowSize;
  
  for (int m=0; m<outputWindowSize; m++)
  {
    int outputIdx = (int)outputIndice_k[m] - 1;
    float outputScale = outputScale_k[m];
    
    for (int j=tx; j<outputSize; j+=i_step)
    {
      buffer[tx] = bias[outputIdx*outputSize + j];
          
      for (int l=0; l<inputWindowSize; l++)
        buffer[tx] += input_k[l*outputWindowSize*outputSize + m*outputSize + j];

      output_k[m*outputSize + j] = outputScale*buffer[tx];
    }
  }
}

static int cunnx_BlockSparse_updateOutput(lua_State *L)
{ 
  /* input, inputIndice, outputIndice, inputScale, outputScale, gradOutput*/
  THCState *state = getCutorchState(L);
  // batchSize x inputWindowSize x inputSize
  THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");  
  // batchSize x inputWindowSize
  THCudaLongTensor *inputIndice = (THCudaLongTensor*)luaT_checkudata(L, 3, "torch.CudaLongTensor");
  THCudaTensor *inputScale = (THCudaTensor*)luaT_checkudata(L, 5, "torch.CudaTensor");
  // batchSize x outputWindowSize
  THCudaLongTensor *outputIndice = (THCudaLongTensor*)luaT_checkudata(L, 4, "torch.CudaLongTensor");
  THCudaTensor *outputScale = (THCudaTensor*)luaT_checkudata(L, 6, "torch.CudaTensor");
  
  int batchSize = luaT_getfieldcheckint(L, 1, "batchSize");
  int inputSize = luaT_getfieldcheckint(L, 1, "inputSize");
  int outputSize = luaT_getfieldcheckint(L, 1, "outputSize");
  int inputWindowSize = luaT_getfieldcheckint(L, 1, "inputWindowSize");
  int outputWindowSize = luaT_getfieldcheckint(L, 1, "outputWindowSize");
  int nInputBlock = luaT_getfieldcheckint(L, 1, "nInputBlock");
  int nOutputBlock = luaT_getfieldcheckint(L, 1, "nOutputBlock");
  int batchedGemmMax = luaT_getfieldcheckint(L, 1, "batchedGemmMax");
  long nBatched = batchSize*inputWindowSize*outputWindowSize;
  
  THLongTensor *inputIndiceHost = (THLongTensor*)luaT_getfieldcheckudata(L, 1, "inputIndiceHost", "torch.LongTensor");
  THLongTensor *outputIndiceHost = (THLongTensor*)luaT_getfieldcheckudata(L, 1, "outputIndiceHost", "torch.LongTensor");
  // nOutputBlock x nInputBlock x outputSize x inputSize
  THCudaTensor *weight = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "weight", "torch.CudaTensor");
  // nOutputBlock x outputSize
  THCudaTensor *bias = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "bias", "torch.CudaTensor");
  // batchSize x inputWindowSize x outputWindowSize x outputSize
  THCudaTensor *outputBatched = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "outputBatched", "torch.CudaTensor");
  // batchSize x outputWindowSize x outputSize
  THCudaTensor *output = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "_output", "torch.CudaTensor");
  
  hipblasStatus_t stat;
  hipblasHandle_t handle;
  
  float alpha = 1;
  float beta = 0;
  
  if (nInputBlock > 1) 
  {
    luaL_argcheck(L, input->nDimension == 3, 2, "3D(batch mode) tensor expected");
    luaL_argcheck(L, input->size[2] == inputSize, 2, "invalid input size"); 
  } 
  else 
  {
    luaL_argcheck(L, input->nDimension == 2, 2, "2D(batch mode) tensor expected");
    luaL_argcheck(L, input->size[1] == inputSize, 2, "invalid input size"); 
  }
  luaL_argcheck(L, inputIndice->nDimension == 2, 3, "2D(batch mode) tensor expected");
  luaL_argcheck(L, outputIndice->nDimension == 2, 4, "2D(batch mode) tensor expected");
  luaL_argcheck(L, inputScale->nDimension == 2, 5, "2D(batch mode) tensor expected");
  luaL_argcheck(L, outputScale->nDimension == 2, 6, "2D(batch mode) tensor expected");
  luaL_argcheck(L, THCudaTensor_isContiguous(state, input), 2, "Expecting contiguous input");
  
  THCudaTensor_resize4d(state, outputBatched, batchSize, inputWindowSize, outputWindowSize, outputSize);
  THLongTensor_resize2d(inputIndiceHost, batchSize, inputWindowSize);
  THLongTensor_resize2d(outputIndiceHost, batchSize, outputWindowSize);
  
  THLongTensor_copyCuda(state, inputIndiceHost, inputIndice);
  THLongTensor_copyCuda(state, outputIndiceHost, outputIndice);
  
  stat = hipblasCreate(&handle);
  if (stat != HIPBLAS_STATUS_SUCCESS) 
    THError("CUBLAS initialization failed");
  
  if ( nOutputBlock > 1 )
    THCudaTensor_resize3d(state, output, batchSize, outputWindowSize, outputSize);
  else
    THCudaTensor_resize2d(state, output, batchSize, outputSize);
  
  /* streamed or batched */
  if (sqrt(inputSize*outputSize) > batchedGemmMax)
  {
    hipStream_t streams[BLOCKSPARSE_STREAMS];
    
    for (int i=0; i<BLOCKSPARSE_STREAMS; i++)
    {
      if (hipStreamCreate(&streams[i]) != hipSuccess)
        THError("error initializing stream");
    }
    hipDeviceSynchronize();
    
    long batchedIdx = 0;
    for (int i=0; i<batchSize; i++)
    {
      float *inputPtr = THCudaTensor_data(state, input)+i*input->stride[0];
      float *outputPtr = THCudaTensor_data(state, outputBatched)+i*outputBatched->stride[0];
      long *inputIdxPtr = THLongTensor_data(inputIndiceHost)+i*inputIndiceHost->stride[0];
      long *outputIdxPtr = THLongTensor_data(outputIndiceHost)+i*outputIndiceHost->stride[0];
      
      for (int l=0; l<inputWindowSize; l++) 
      {              
        for (int m=0; m<outputWindowSize; m++)
        {
          hipblasSetStream(handle, streams[batchedIdx%BLOCKSPARSE_STREAMS]);
      
          stat = hipblasSgemv(handle, HIPBLAS_OP_T,  inputSize, outputSize,
                            &alpha, (const float*)THCudaTensor_data(state, weight)+(inputIdxPtr[l]-1)*weight->stride[1] + (outputIdxPtr[m]-1)*weight->stride[0], inputSize,
                            (const float*)inputPtr, 1,
                            &beta, outputPtr, 1);
                            
          if (stat != HIPBLAS_STATUS_SUCCESS) 
            THError("hipblasSgemv failed");

          outputPtr += outputBatched->stride[2];
          batchedIdx++;
        }
        
        inputPtr += input->stride[1];
      }
    }
    
    hipblasSetStream(handle, NULL);
    hipDeviceSynchronize();
    
    for (int i=0; i<BLOCKSPARSE_STREAMS; i++)
    {
      if (hipStreamDestroy(streams[i]) != hipSuccess)
        THError("error destroying stream");
    }
    
  }
  else
  {  
    THCharTensor *inputHost = (THCharTensor*)luaT_getfieldcheckudata(L, 1, "inputHost", "torch.CharTensor");
    THCharTensor *weightHost = (THCharTensor*)luaT_getfieldcheckudata(L, 1, "weightHost", "torch.CharTensor");
    THCharTensor *outputHost = (THCharTensor*)luaT_getfieldcheckudata(L, 1, "outputHost", "torch.CharTensor");
    
    THCudaTensor *inputCuda = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "inputCuda", "torch.CudaTensor");
    THCudaTensor *weightCuda = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "weightCuda", "torch.CudaTensor");
    THCudaTensor *outputCuda = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "outputCuda", "torch.CudaTensor");
  
    // put output back on top of the stack
    output = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "_output", "torch.CudaTensor");
    
    hipblasSetStream(handle, NULL);
    
    THCharTensor_resize1d(inputHost, nBatched*sizeof(float*));
    THCharTensor_resize1d(weightHost, nBatched*sizeof(float*));
    THCharTensor_resize1d(outputHost, nBatched*sizeof(float*));
    
    THCudaTensor_resize1d(state, inputCuda, nBatched*sizeof(float*)/sizeof(float));
    THCudaTensor_resize1d(state, weightCuda, nBatched*sizeof(float*)/sizeof(float));
    THCudaTensor_resize1d(state, outputCuda, nBatched*sizeof(float*)/sizeof(float));
    
    const float **inputB = (const float **)THCharTensor_data(inputHost);
    const float **weightB = (const float **)THCharTensor_data(weightHost);
    float **outputB = (float **)THCharTensor_data(outputHost);
    
    const float **inputB_d = (const float **)THCudaTensor_data(state, inputCuda);
    const float **weightB_d = (const float **)THCudaTensor_data(state, weightCuda);
    float **outputB_d = (float **)THCudaTensor_data(state, outputCuda);
    
    long batchedIdx = 0;
    for (int i=0; i<batchSize; i++)
    {
      float *inputPtr = THCudaTensor_data(state, input)+i*input->stride[0];
      float *outputPtr = THCudaTensor_data(state, outputBatched)+i*outputBatched->stride[0];
      long *inputIdxPtr = THLongTensor_data(inputIndiceHost)+i*inputIndiceHost->stride[0];
      long *outputIdxPtr = THLongTensor_data(outputIndiceHost)+i*outputIndiceHost->stride[0];
      
      for (int l=0; l<inputWindowSize; l++) 
      {              
        for (int m=0; m<outputWindowSize; m++)
        {
          inputB[batchedIdx] = inputPtr;
          weightB[batchedIdx] = THCudaTensor_data(state, weight) + (outputIdxPtr[m]-1)*weight->stride[0] + (inputIdxPtr[l]-1)*weight->stride[1];
          outputB[batchedIdx] = outputPtr;

          outputPtr += outputBatched->stride[2];
          batchedIdx++;
        }
        
        inputPtr += input->stride[1];
      }
    }
    
    if(hipMemcpy(inputB_d, inputB, sizeof(float*) * nBatched, hipMemcpyHostToDevice) != hipSuccess)
      THError("hipMemcpy failed");
    if(hipMemcpy(weightB_d, weightB, sizeof(float*) * nBatched, hipMemcpyHostToDevice) != hipSuccess)
      THError("hipMemcpy failed");
    if(hipMemcpy(outputB_d, outputB, sizeof(float*) * nBatched, hipMemcpyHostToDevice) != hipSuccess)
      THError("hipMemcpy failed");
    
    stat = hipblasSgemmBatched(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                             outputSize, 1, inputSize,
                             &alpha, weightB_d, inputSize, 
                             inputB_d, inputSize, 
                             &beta, outputB_d, outputSize, 
                             nBatched);
    
    if (stat != HIPBLAS_STATUS_SUCCESS) 
      THError("hipblasSgemmBatched failed");
    
  }
  
  /* call cudakernel */
  dim3 blocks(input->size[0]); // each cuda-block is an example
  dim3 threads(BLOCKSPARSE_THREADS);
  cunnx_BlockSparse_updateOutput_kernel<<<blocks,threads>>>(
    THCudaTensor_data(state, output), THCudaTensor_data(state, outputBatched), 
    (const float *)THCudaLongTensor_data(state, outputIndice), THCudaTensor_data(state, outputScale),
    THCudaTensor_data(state, bias),  outputSize, nOutputBlock,
    inputWindowSize, outputWindowSize
  );
  
  hipblasDestroy(handle);
  
  hipError_t errcode = hipGetLastError();
  if(errcode != hipSuccess)
    THError(hipGetErrorString(errcode));

  return 1;
}
  
__global__ void cunnx_BlockSparse_updateGradOutput_kernel(
  float *_gradOutput, float* gradOutputScale, const float *gradOutput, 
  const float *output, const float *outputScale, 
  int outputWindowSize, int outputSize)
{
  __shared__ float buffer[BLOCKSPARSE_THREADS];
  int tx = threadIdx.x;
  int i_step = blockDim.x;
  int k = blockIdx.x;
  
  float *_gradOutput_k = _gradOutput + k*outputWindowSize*outputSize;
  float *gradOutputScale_k = gradOutputScale + k*outputWindowSize;
  const float *gradOutput_k = gradOutput + k*outputWindowSize*outputSize;
  const float *output_k = output + k*outputWindowSize*outputSize;
  const float *outputScale_k = outputScale + k*outputWindowSize;
  
  
  // get gradients for outputScale (to be backwarded to a Gater)
  for (int m=0; m<outputWindowSize; m++)
  {
    float outputScale = outputScale_k[m];
    
    float *_blockGradOutput = _gradOutput_k + m*outputSize;  
    const float *blockGradOutput = gradOutput_k + m*outputSize;
    const float *blockOutput = output_k + m*outputSize;
    
    buffer[tx] = 0;
    
    for (int j=tx; j<outputSize; j+=i_step)
    {
      const float grad = blockGradOutput[j];
      buffer[tx] += blockOutput[j]*grad;
      _blockGradOutput[j] = grad*outputScale;
    }
    
    // add (reduce)
    for (unsigned int stride = blockDim.x >> 1; stride > 0; stride >>= 1)
    {
      __syncthreads();
      if (tx < stride)
        buffer[tx] += buffer[tx+stride];
    }
    
    if (tx == 0)
      gradOutputScale_k[m] = buffer[0]/(outputScale+0.00000001);
  }
}


static int cunnx_BlockSparse_updateGradInput(lua_State *L)
{   
  /* input, inputIndice, outputIndice, inputScale, outputScale*/
  THCState *state = getCutorchState(L);
  // batchSize x inputWindowSize x inputSize
  THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");  
  // batchSize x inputWindowSize
  THCudaTensor *inputIndice = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
  THCudaTensor *inputScale = (THCudaTensor*)luaT_checkudata(L, 5, "torch.CudaTensor");
  // batchSize x outputWindowSize
  THCudaTensor *outputIndice = (THCudaTensor*)luaT_checkudata(L, 4, "torch.CudaTensor");
  THCudaTensor *outputScale = (THCudaTensor*)luaT_checkudata(L, 6, "torch.CudaTensor");
  // batchSize x outputWindowSize x outputSize
  THCudaTensor *gradOutput = (THCudaTensor*)luaT_checkudata(L, 7, "torch.CudaTensor");
  THCudaTensor *output = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "_output", "torch.CudaTensor");
  
  int batchSize = luaT_getfieldcheckint(L, 1, "batchSize");
  int inputSize = luaT_getfieldcheckint(L, 1, "inputSize");
  int outputSize = luaT_getfieldcheckint(L, 1, "outputSize");
  int inputWindowSize = luaT_getfieldcheckint(L, 1, "inputWindowSize");
  int outputWindowSize = luaT_getfieldcheckint(L, 1, "outputWindowSize");
  int nInputBlock = luaT_getfieldcheckint(L, 1, "nInputBlock");
  int nOutputBlock = luaT_getfieldcheckint(L, 1, "nOutputBlock");
  int batchedGemmMax = luaT_getfieldcheckint(L, 1, "batchedGemmMax");
  long nBatched = batchSize*inputWindowSize*outputWindowSize;
  
  THLongTensor *inputIndiceHost = (THLongTensor*)luaT_getfieldcheckudata(L, 1, "inputIndiceHost", "torch.LongTensor");
  THLongTensor *outputIndiceHost = (THLongTensor*)luaT_getfieldcheckudata(L, 1, "outputIndiceHost", "torch.LongTensor");
  THCudaTensor *weight = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "weight", "torch.CudaTensor");
  THCudaTensor *gradInputBatched = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "gradInputBatched", "torch.CudaTensor");
  THCudaTensor *_gradOutput = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "_gradOutput", "torch.CudaTensor");
  THCudaTensor *gradInput = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "_gradInput", "torch.CudaTensor");
  THCudaTensor *gradOutputScale = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "gradOutputScale", "torch.CudaTensor");
  
  hipblasStatus_t stat;
  hipblasHandle_t handle;
  
  float alpha = 1;
  float beta = 0;
  
  if (nInputBlock > 1) 
  {
    luaL_argcheck(L, input->nDimension == 3, 2, "3D(batch mode) tensor expected");
    luaL_argcheck(L, input->size[2] == inputSize, 2, "invalid input size"); 
  } 
  else 
  {
    luaL_argcheck(L, input->nDimension == 2, 2, "2D(batch mode) tensor expected");
    luaL_argcheck(L, input->size[1] == inputSize, 2, "invalid input size"); 
  }
  luaL_argcheck(L, inputIndice->nDimension == 2, 3, "2D(batch mode) tensor expected");
  luaL_argcheck(L, outputIndice->nDimension == 2, 4, "2D(batch mode) tensor expected");
  luaL_argcheck(L, inputScale->nDimension == 2, 5, "2D(batch mode) tensor expected");
  luaL_argcheck(L, outputScale->nDimension == 2, 6, "2D(batch mode) tensor expected");
  luaL_argcheck(L, THCudaTensor_isContiguous(state, input), 2, "Expecting contiguous input");
  
  THCudaTensor_resizeAs(state, _gradOutput, gradOutput);
  THCudaTensor_resizeAs(state, gradOutputScale, outputScale);
  THCudaTensor_resize4d(state, gradInputBatched, batchSize, outputWindowSize, inputWindowSize, inputSize);
 
  /* call cudakernel */
  dim3 blocks(input->size[0]); // each cuda-block is an example
  dim3 threads(BLOCKSPARSE_THREADS);
  cunnx_BlockSparse_updateGradOutput_kernel<<<blocks,threads>>>(
    THCudaTensor_data(state, _gradOutput), THCudaTensor_data(state, gradOutputScale), 
    THCudaTensor_data(state, gradOutput), THCudaTensor_data(state, output),
    THCudaTensor_data(state, outputScale), outputWindowSize, outputSize
  );
  
  hipError_t errcode = hipGetLastError();
  if(errcode != hipSuccess)
    THError(hipGetErrorString(errcode));
    
  stat = hipblasCreate(&handle);
  if (stat != HIPBLAS_STATUS_SUCCESS) 
    THError("CUBLAS initialization failed");
  
  /* streamed or batched */
  if (sqrt(inputSize*outputSize) > batchedGemmMax)
  {
    hipStream_t streams[BLOCKSPARSE_STREAMS];
    
    for (int i=0; i<BLOCKSPARSE_STREAMS; i++)
    {
      if (hipStreamCreate(&streams[i]) != hipSuccess)
        THError("error initializing stream");
    }
    hipDeviceSynchronize();
    
    long batchedIdx = 0;
    for (int i=0; i<batchSize; i++)
    {
      float *gradOutputPtr = THCudaTensor_data(state, _gradOutput)+i*_gradOutput->stride[0];
      float *gradInputPtr = THCudaTensor_data(state, gradInputBatched)+i*gradInputBatched->stride[0];
      long *inputIdxPtr = THLongTensor_data(inputIndiceHost)+i*inputIndiceHost->stride[0];
      long *outputIdxPtr = THLongTensor_data(outputIndiceHost)+i*outputIndiceHost->stride[0];
      
      for (int m=0; m<outputWindowSize; m++)
      {              
        for (int l=0; l<inputWindowSize; l++) 
        {
          hipblasSetStream(handle, streams[batchedIdx%BLOCKSPARSE_STREAMS]);
      
          stat = hipblasSgemv(handle, HIPBLAS_OP_N,  inputSize, outputSize,
                            &alpha, (const float*)THCudaTensor_data(state, weight)+(outputIdxPtr[m]-1)*weight->stride[0]+(inputIdxPtr[l]-1)*weight->stride[1], inputSize,
                            (const float*)gradOutputPtr, 1,
                            &beta, gradInputPtr, 1);
                            
          if (stat != HIPBLAS_STATUS_SUCCESS) 
            THError("hipblasSgemv failed");

          gradInputPtr += gradInputBatched->stride[2];
          batchedIdx++;
        }
        
        gradOutputPtr += _gradOutput->stride[1];
      }
    }
    
    hipblasSetStream(handle, NULL);
    hipDeviceSynchronize();
    
    for (int i=0; i<BLOCKSPARSE_STREAMS; i++)
    {
      if (hipStreamDestroy(streams[i]) != hipSuccess)
        THError("error destroying stream");
    }
    
  }
  else
  {  
    THCharTensor *inputHost = (THCharTensor*)luaT_getfieldcheckudata(L, 1, "inputHost", "torch.CharTensor");
    THCharTensor *weightHost = (THCharTensor*)luaT_getfieldcheckudata(L, 1, "weightHost", "torch.CharTensor");
    THCharTensor *outputHost = (THCharTensor*)luaT_getfieldcheckudata(L, 1, "outputHost", "torch.CharTensor");
    
    THCudaTensor *inputCuda = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "inputCuda", "torch.CudaTensor");
    THCudaTensor *weightCuda = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "weightCuda", "torch.CudaTensor");
    THCudaTensor *outputCuda = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "outputCuda", "torch.CudaTensor");
    // put gradInput back on top of the stack
    gradInput = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "_gradInput", "torch.CudaTensor");
    gradOutputScale = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "gradOutputScale", "torch.CudaTensor");
    
    hipblasSetStream(handle, NULL);
    
    THCharTensor_resize1d(inputHost, nBatched*sizeof(float*));
    THCharTensor_resize1d(weightHost, nBatched*sizeof(float*));
    THCharTensor_resize1d(outputHost, nBatched*sizeof(float*));
    
    THCudaTensor_resize1d(state, inputCuda, nBatched*sizeof(float*)/sizeof(float));
    THCudaTensor_resize1d(state, weightCuda, nBatched*sizeof(float*)/sizeof(float));
    THCudaTensor_resize1d(state, outputCuda, nBatched*sizeof(float*)/sizeof(float));
    
    float **gradInputB = (float **)THCharTensor_data(inputHost);
    const float **weightB = (const float **)THCharTensor_data(weightHost);
    const float **gradOutputB = (const float **)THCharTensor_data(outputHost);
    
    float **gradInputB_d = (float **)THCudaTensor_data(state, inputCuda);
    const float **weightB_d = (const float **)THCudaTensor_data(state, weightCuda);
    const float **gradOutputB_d = (const float **)THCudaTensor_data(state, outputCuda);
    

    long batchedIdx = 0;
    for (int i=0; i<batchSize; i++)
    {
      float *gradOutputPtr = THCudaTensor_data(state, _gradOutput)+i*_gradOutput->stride[0];
      float *gradInputPtr = THCudaTensor_data(state, gradInputBatched)+i*gradInputBatched->stride[0];
      long *inputIdxPtr = THLongTensor_data(inputIndiceHost)+i*inputIndiceHost->stride[0];
      long *outputIdxPtr = THLongTensor_data(outputIndiceHost)+i*outputIndiceHost->stride[0];
      
      for (int m=0; m<outputWindowSize; m++)
      {              
        for (int l=0; l<inputWindowSize; l++) 
        {
          gradInputB[batchedIdx] = gradInputPtr;
          weightB[batchedIdx] = THCudaTensor_data(state, weight)+(outputIdxPtr[m]-1)*weight->stride[0]+(inputIdxPtr[l]-1)*weight->stride[1];
          gradOutputB[batchedIdx] = gradOutputPtr;

          gradInputPtr += gradInputBatched->stride[2];
          batchedIdx++;
        }
        
        gradOutputPtr += _gradOutput->stride[1];
      }
    }
    
    if(hipMemcpy(gradInputB_d, gradInputB, sizeof(float*)*nBatched, hipMemcpyHostToDevice) != hipSuccess)
      THError("hipMemcpy failed");
    if(hipMemcpy(weightB_d, weightB, sizeof(float*)*nBatched, hipMemcpyHostToDevice) != hipSuccess)
      THError("hipMemcpy failed");
    if(hipMemcpy(gradOutputB_d, gradOutputB, sizeof(float*)*nBatched, hipMemcpyHostToDevice) != hipSuccess)
      THError("hipMemcpy failed");

    stat = hipblasSgemmBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                             inputSize, 1, outputSize,
                             &alpha, weightB_d, inputSize, 
                             gradOutputB_d, outputSize, 
                             &beta, gradInputB_d, inputSize, 
                             nBatched);
    
    if (stat != HIPBLAS_STATUS_SUCCESS) 
      THError("hipblasSgemmBatched failed");
    
  }
  
  hipblasDestroy(handle);
  
  THCudaTensor_sum(state, gradInput, gradInputBatched, 0, 1);
  THCudaTensor_resizeAs(state, gradInput, input); 
  
  errcode = hipGetLastError();
  if(errcode != hipSuccess)
    THError(hipGetErrorString(errcode));

  return 2;
}
  
__global__ void cunnx_BlockSparse_accGradParameters_kernel(
  float *gradWeight, float* gradBias, float *gradOutput, 
  float *input, float *inputIndice, float *outputIndice, 
  int inputSize, int outputSize, int nInputBlock, int nOutputBlock,
  int inputWindowSize, int outputWindowSize, float scale)
{
  __shared__ float buffer[BLOCKSPARSE_THREADS];
  __shared__ float gradOutputBuffer[BLOCKSPARSE_MAXOUTPUTBLOCKSIZE];
  int tx = threadIdx.x;
  int i_step = blockDim.x;
  int k = blockIdx.x;
  
  float *input_k = input + k*inputWindowSize*inputSize;
  float *gradOutput_k = gradOutput + k*outputWindowSize*outputSize;
  float *inputIndice_k = inputIndice + k*inputWindowSize;
  float *outputIndice_k = outputIndice + k*outputWindowSize;
  
  // loop through blocks
  for (int m=0; m<outputWindowSize; m++)
  {
    int outputIdx = (int)outputIndice_k[m] - 1;
      
    float *blockGradOutput = gradOutput_k + m*outputSize;
    float *blockGradBias = gradBias + outputIdx*outputSize;
    
    for (int j=tx; j<outputSize; j+=i_step)
      gradOutputBuffer[j] = blockGradOutput[j]*scale;
    
    __syncthreads(); // needed for some reason
    
    for (int l=0; l<inputWindowSize; l++)
    {
      int inputIdx = (int)inputIndice_k[l] - 1;
      
      float *blockInput = input_k + l*inputSize;
      float *blockGradWeight = gradWeight + outputIdx*nInputBlock*outputSize*inputSize + inputIdx*outputSize*inputSize;
      
      // addr weights (scalar-products)
      for (int i=tx; i<inputSize; i+=i_step)
      {
        // copy input to buffer
        buffer[tx] = blockInput[i];
      
        // multiply accumulate weights
        for (int j=0; j<outputSize; j++)
          atomicAdd(&(blockGradWeight[j*inputSize + i]), gradOutputBuffer[j]*buffer[tx]);
      }
    }
    
    __syncthreads(); // needed for some reason
    
    // multiply accumulate biases 
    for (int j=tx; j<outputSize; j+=i_step)
      atomicAdd(&(blockGradBias[j]), gradOutputBuffer[j]);
  }
}


static int cunnx_BlockSparse_accGradParameters(lua_State *L)
{ 
  /* input, inputIndice, outputIndice, inputScale, outputScale, gradOutput, scale */
  THCState *state = getCutorchState(L);
  // batchSize x inputWindowSize x inputSize
  THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");  
  // batchSize x inputWindowSize
  THCudaTensor *inputIndice = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
  THCudaTensor *inputScale = (THCudaTensor*)luaT_checkudata(L, 5, "torch.CudaTensor");
  // batchSize x outputWindowSize
  THCudaTensor *outputIndice = (THCudaTensor*)luaT_checkudata(L, 4, "torch.CudaTensor");
  THCudaTensor *outputScale = (THCudaTensor*)luaT_checkudata(L, 6, "torch.CudaTensor");
  float scale = luaL_optnumber(L, 8, 1);
  
  int inputSize = luaT_getfieldcheckint(L, 1, "inputSize");
  int outputSize = luaT_getfieldcheckint(L, 1, "outputSize");
  int nInputBlock = luaT_getfieldcheckint(L, 1, "nInputBlock");
  int nOutputBlock = luaT_getfieldcheckint(L, 1, "nOutputBlock");
  
  THCudaTensor *gradWeight = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "gradWeight", "torch.CudaTensor");
  THCudaTensor *gradBias = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "gradBias", "torch.CudaTensor");
  THCudaTensor *_gradOutput = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "_gradOutput", "torch.CudaTensor");
  THLongTensor *inputIndiceHost = (THLongTensor*)luaT_getfieldcheckudata(L, 1, "inputIndiceHost", "torch.LongTensor");
  THLongTensor *outputIndiceHost = (THLongTensor*)luaT_getfieldcheckudata(L, 1, "outputIndiceHost", "torch.LongTensor");
  
  if (nInputBlock > 1) 
  {
    luaL_argcheck(L, input->nDimension == 3, 2, "3D(batch mode) tensor expected");
    luaL_argcheck(L, input->size[2] == inputSize, 2, "invalid input size"); 
  } 
  else 
  {
    luaL_argcheck(L, input->nDimension == 2, 2, "2D(batch mode) tensor expected");
    luaL_argcheck(L, input->size[1] == inputSize, 2, "invalid input size"); 
  }
  luaL_argcheck(L, inputIndice->nDimension == 2, 3, "2D(batch mode) tensor expected");
  luaL_argcheck(L, outputIndice->nDimension == 2, 4, "2D(batch mode) tensor expected");
  luaL_argcheck(L, inputScale->nDimension == 2, 5, "2D(batch mode) tensor expected");
  luaL_argcheck(L, outputScale->nDimension == 2, 6, "2D(batch mode) tensor expected");
  luaL_argcheck(L, outputSize <= BLOCKSPARSE_MAXOUTPUTBLOCKSIZE, 1, "outputSize is too large");
  
  /* call cudakernel */
  dim3 blocks(input->size[0]); // each cuda-block is an example
  dim3 threads(BLOCKSPARSE_THREADS);
  cunnx_BlockSparse_accGradParameters_kernel<<<blocks,threads>>>(
    THCudaTensor_data(state, gradWeight), THCudaTensor_data(state, gradBias), 
    THCudaTensor_data(state, _gradOutput), THCudaTensor_data(state, input),
    THCudaTensor_data(state, inputIndice), THCudaTensor_data(state, outputIndice), 
    inputSize, outputSize, nInputBlock, nOutputBlock, 
    inputIndice->size[1], outputIndice->size[1], scale
  );
  
  hipError_t errcode = hipGetLastError();
  if(errcode != hipSuccess)
    THError(hipGetErrorString(errcode));
  
  return 0;
}


  
static const struct luaL_Reg cunnx_BlockSparse__ [] = {
  {"BlockSparse_updateOutput", cunnx_BlockSparse_updateOutput},
  {"BlockSparse_updateGradInput", cunnx_BlockSparse_updateGradInput},
  {"BlockSparse_accGradParameters", cunnx_BlockSparse_accGradParameters},
  {NULL, NULL}
};

static void cunnx_BlockSparse_init(lua_State *L)
{
  luaT_pushmetatable(L, "torch.CudaTensor");
  luaT_registeratname(L, cunnx_BlockSparse__, "nn");
  lua_pop(L,1);
}
